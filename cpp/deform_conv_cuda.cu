#include "hip/hip_runtime.h"
#include <vector>
#include "deform_im2col_cuda.cuh"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// #include <THC/THC.h>
// #include <THC/THCAtomics.cuh>
// #include <THC/THCDeviceUtils.cuh>

// extern THCState *state;

// author: Charles Shang
// https://github.com/torch/cunn/blob/master/lib/THCUNN/generic/SpatialConvolutionMM.cu


at::Tensor
deform_conv_cuda_forward(const at::Tensor &input,
                    const at::Tensor &weight,
                    const at::Tensor &bias,
                    const int kernel_h,
                    const int kernel_w,
                    const int stride_h,
                    const int stride_w,
                    const int pad_h,
                    const int pad_w,
                    const int dilation_h,
                    const int dilation_w,
                    const int groups,
                    const int im2col_step)
{
    // THCAssertSameGPU(THCudaTensor_checkGPU(state, 5, input, weight, bias, offset, mask));

    AT_ASSERTM(input.is_contiguous(), "input tensor has to be contiguous");
    AT_ASSERTM(weight.is_contiguous(), "weight tensor has to be contiguous");
    AT_ASSERTM(bias.is_contiguous(), "bias tensor has to be contiguous");

    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(weight.type().is_cuda(), "weight must be a CUDA tensor");
    AT_ASSERTM(bias.type().is_cuda(), "bias must be a CUDA tensor");

    const int batch = input.size(0);
    const int channels = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int channels_kernel = weight.size(1);
    const int kernel_h_ = weight.size(2);
    const int kernel_w_ = weight.size(3);

    const int im2col_step_ = std::min(batch, im2col_step);

    AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)", batch, im2col_step_);

    AT_ASSERTM((channels % groups == 0) && (channels_out % groups == 0), 
        "channels(%d) and channels_out(%d) must divide groups(%d)", channels, channels_out, groups);

    // printf("Kernels: %d %d %d %d\n", kernel_h_, kernel_w_, kernel_w, kernel_h);
    // printf("Channels: %d %d\n", channels, channels_kernel);
    // printf("Channels: %d %d\n", channels_out, channels_kernel);

    AT_ASSERTM(kernel_h_ == kernel_h && kernel_w_ == kernel_w,
               "Input shape and kernel shape wont match: (%d x %d vs %d x %d).", kernel_h_, kernel_w, kernel_h_, kernel_w_);

    AT_ASSERTM(channels == (channels_kernel * groups),
               "Input shape and kernel channels wont match: (%d vs %d).", channels, channels_kernel * groups);

    const int height_out = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

    auto output = at::empty({batch * height_out * width_out, channels_out}, input.options());

    // prepare groups weight and bias
    auto weight_g = weight.view({groups, channels_out/groups, channels_kernel, kernel_h, kernel_w});
    auto bias_g = bias.view({groups, channels_out/groups});

    // define alias for easy use
    const int batch_n = im2col_step_;
    const int per_input_size = channels * height * width;
    auto output_n = output.view({batch/im2col_step_, batch_n * height_out * width_out, channels_out});
    for (int n = 0; n < batch/im2col_step_; ++n)
    {
        auto columns = at::empty({channels * kernel_h * kernel_w, batch_n * height_out * width_out}, input.options());
        AT_DISPATCH_FLOATING_TYPES(input.type(), "deform_conv_forward_cuda", ([&] {
            deformable_im2col_cuda(at::cuda::getCurrentCUDAStream(),
                                             input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                                             batch_n, channels, 
                                             height, width,
                                             kernel_h, kernel_w,
                                             stride_h, stride_w, 
                                             pad_h, pad_w, 
                                             dilation_h, dilation_w,
                                             height_out, width_out, 
                                             columns.data<scalar_t>());

        }));

        // auto columns_m = columns.t();
        // auto weight_m = weight.view({channels_out, channels_kernel * kernel_h * kernel_w}).t();
        // output = at::addmm(bias, columns_m, weight_m);
        auto columns_g = columns.view({groups, channels/groups * kernel_h * kernel_w, batch_n * height_out * width_out});
        auto output_g = output_n.select(0, n).view({batch_n * height_out * width_out, groups, channels_out/groups});
        for (int g = 0; g < groups; ++g)
        {
            auto columns_gm = columns_g.select(0, g).t();
            auto weight_gm = weight_g.select(0, g).view({channels_out/groups, channels_kernel * kernel_h * kernel_w}).t();
            auto output_m = at::addmm(bias_g.select(0, g), columns_gm, weight_gm);
            output_g.select(1, g) = output_m.view({batch_n * height_out * width_out, channels_out/groups});
        }

    }

    output = output.view({batch, height_out, width_out, channels_out}).permute({0, 3, 1, 2}).contiguous();

    return output;
}

std::vector<at::Tensor> deform_conv_cuda_backward(const at::Tensor &input,
                                             const at::Tensor &weight,
                                             const at::Tensor &bias,
                                             const at::Tensor &grad_output,
                                             const int kernel_h, 
                                             const int kernel_w,
                                             const int stride_h, 
                                             const int stride_w,
                                             const int pad_h, 
                                             const int pad_w,
                                             const int dilation_h, 
                                             const int dilation_w,
                                             const int groups,
                                             const int im2col_step)
{

    AT_ASSERTM(input.is_contiguous(), "input tensor has to be contiguous");
    AT_ASSERTM(weight.is_contiguous(), "weight tensor has to be contiguous");
    AT_ASSERTM(bias.is_contiguous(), "weight tensor has to be contiguous");

    AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
    AT_ASSERTM(weight.type().is_cuda(), "weight must be a CUDA tensor");
    AT_ASSERTM(bias.type().is_cuda(), "bias must be a CUDA tensor");

    const int batch = input.size(0);
    const int channels = input.size(1);
    const int height = input.size(2);
    const int width = input.size(3);

    const int channels_out = weight.size(0);
    const int channels_kernel = weight.size(1);
    const int kernel_h_ = weight.size(2);
    const int kernel_w_ = weight.size(3);

    const int batch_ = grad_output.size(0);
    const int channels_out_ = grad_output.size(1);
    const int height_out_ = grad_output.size(2);
    const int width_out_ = grad_output.size(3);

    const int im2col_step_ = std::min(im2col_step, batch);

    AT_ASSERTM(batch % im2col_step_ == 0, "batch(%d) must divide im2col_step(%d)", batch, im2col_step_);

    AT_ASSERTM((channels % groups == 0) && (channels_out % groups == 0), 
        "channels(%d) and channels_out(%d) must divide groups(%d)", channels, channels_out, groups);

    AT_ASSERTM(kernel_h_ == kernel_h && kernel_w_ == kernel_w,
               "Input shape and kernel shape wont match: (%d x %d vs %d x %d).", kernel_h_, kernel_w, kernel_h_, kernel_w_);

    AT_ASSERTM(channels == (channels_kernel * groups),
               "Input shape and kernel channels wont match: (%d vs %d).", channels, channels_kernel * groups);

    const int height_out = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int width_out = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;

    AT_ASSERTM(batch == batch_,
               "Input shape and grad_out batch wont match: (%d vs %d).", batch, batch_);

    AT_ASSERTM(channels_out == channels_out_,
               "Input shape and grad_out channels_out wont match: (%d vs %d).", channels_out, channels_out_);

    AT_ASSERTM(height_out == height_out_ && width_out == width_out_,
               "Input shape and grad_out shape wont match: (%d x %d vs %d x %d).", height_out, height_out_, width_out, width_out_);

    auto grad_input = at::zeros_like(input);
    auto grad_weight = at::zeros_like(weight);
    auto grad_bias = at::zeros_like(bias);

    // auto grad_output_m = grad_output.permute({1, 0, 2, 3}).contiguous().view({channels_out, batch * height_out * width_out});
    // auto weight_m = weight.view({channels_out, channels_kernel * kernel_h * kernel_w}).t();
    // columns = at::mm(weight_m, grad_output_m);

    // prepare groups weight and bias
    auto weight_g = weight.view({groups, channels_out/groups, channels_kernel, kernel_h, kernel_w});
    auto grad_weight_g = grad_weight.view({groups, channels_out/groups, channels_kernel, kernel_h, kernel_w});
    auto grad_bias_g = grad_bias.view({groups, channels_out/groups});

    const int batch_n = im2col_step_;
    const int per_input_size = channels * height * width;
    auto grad_output_n = grad_output.view({batch/im2col_step_, batch_n, channels_out, height_out, width_out});
    for (int n = 0; n < batch/im2col_step_; ++n)
    {
        auto ones = at::ones({batch_n * height_out * width_out}, input.options());
        auto grad_output_g = grad_output_n.select(0, n).view({batch_n, groups, channels_out/groups, height_out, width_out});
        auto columns = at::empty({channels * kernel_h * kernel_w, batch_n * 1 * height_out * width_out}, input.options());
        auto columns_g = columns.view({groups, channels/groups * kernel_h * kernel_w, batch_n * height_out * width_out});
        for (int g = 0; g < groups; ++g)
        {
            auto grad_output_gm = grad_output_g.select(1, g).permute({1, 0, 2, 3}).contiguous().view({channels_out/groups, batch_n * height_out * width_out});
            auto weight_gm = weight_g.select(0, g).view({channels_out/groups, channels_kernel * kernel_h * kernel_w}).t();
            columns_g.select(0, g) = at::mm(weight_gm, grad_output_gm);
        }

        AT_DISPATCH_FLOATING_TYPES(input.type(), "deform_conv_backward_cuda", ([&] {
            // gradient w.r.t. input data
            deformable_col2im_cuda(at::cuda::getCurrentCUDAStream(),
                                             columns.data<scalar_t>(),
                                             batch_n, channels, 
                                             height, width,
                                             kernel_h, kernel_w,
                                             stride_h, stride_w,
                                             pad_h, pad_w, 
                                             dilation_h, dilation_w,
                                             height_out, width_out, 
                                             grad_input.data<scalar_t>() + n * im2col_step_ * per_input_size);

            // gradient w.r.t. weight, dWeight should accumulate across the batch and groups
            deformable_im2col_cuda(at::cuda::getCurrentCUDAStream(),
                                             input.data<scalar_t>() + n * im2col_step_ * per_input_size,
                                             batch_n, channels, 
                                             height, width,
                                             kernel_h, kernel_w,
                                             stride_h, stride_w,
                                             pad_h, pad_w, 
                                             dilation_h, dilation_w,
                                             height_out, width_out, 
                                             columns.data<scalar_t>());

        }));

        // auto grad_output_m = grad_output.permute({1, 0, 2, 3}).contiguous().view({channels_out, batch * height_out * width_out});
        // grad_weight = at::mm(grad_output_m, columns.t()).view_as(weight);
        // grad_bias = at::mv(grad_output_m, ones);
        // auto grad_output_g = grad_output.view({batch, groups, channels_out/groups, height_out, width_out});
        // auto columns_g = columns.view({groups, channels/groups * kernel_h * kernel_w, batch * height_out * width_out});
        for (int g = 0; g < groups; ++g)
        {
            auto grad_output_gm = grad_output_g.select(1, g).permute({1, 0, 2, 3}).contiguous().view({channels_out/groups, batch_n * height_out * width_out});
            auto columns_gm = columns_g.select(0, g).t();
            auto grad_weight_gm = grad_weight_g.select(0, g).view({channels_out/groups, channels_kernel * kernel_h * kernel_w});
            auto grad_bias_gm = grad_bias_g.select(0, g);
            grad_weight_g.select(0, g) = at::addmm(grad_weight_gm, grad_output_gm, columns_gm).view_as(grad_weight_g.select(0, g));
            grad_bias_g.select(0, g) = at::addmv(grad_bias_gm, grad_output_gm, ones);
        }

    }

    return {
        grad_input, grad_weight, grad_bias
    };
}
